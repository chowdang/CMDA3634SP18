#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

__device__ unsigned int kmodprod(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

__device__ unsigned int kmodExp(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = kmodprod(aExpb, z, p);
    z = kmodprod(z, z, p);
    b /= 2;
  }
  return aExpb;
}


__global__ void madan(unsigned int *p, unsigned int *g, unsigned int *h, unsigned int *x) {
   int blockid = blockIdx.x;
   int threadid = threadIdx.x;
   int Nblock = blockDim.x;
   
   int id = threadid + blockid * Nblock;
   if (id < *p - 1) {

    if (kmodExp(*g, id + 1, *p) == *h) {

       *x = id + 1;
       printf("Secret Key Found! x = %u \n", id + 1);
  
    } 
   }
}
int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
  FILE * m = fopen("message.txt", "r");
  FILE * pk = fopen("public_key.txt", "r");

  fscanf(pk, "%d\n%d\n%d\n%d\n", &n, &p, &g, &h);
  fscanf(m, "%d\n", &Nints);

  unsigned int charsPerInt = (n - 1) / 8;
  unsigned int *Zmessage =
      (unsigned int *) malloc(Nints*sizeof(unsigned int));
  unsigned int *a =
      (unsigned int *) malloc(Nints*sizeof(unsigned int));
  for (unsigned int i = 0; i<Nints; i++) {
    fscanf(m, "%d %d\n", &Zmessage[i], &a[i]);
  }

  fclose(pk);
  fclose(m);
  // find the secret key
  unsigned int *d_p, *d_g, *d_h, *d_x;

  hipMalloc(&d_p, sizeof(unsigned int));
  hipMalloc(&d_g, sizeof(unsigned int));
  hipMalloc(&d_h, sizeof(unsigned int));
  hipMalloc(&d_x, sizeof(unsigned int));

  hipMemcpy(d_p, &p, sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(d_g, &g, sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(d_h, &h, sizeof(unsigned int), hipMemcpyHostToDevice);

  int Nthreads = 1024;
  int Nblocks = (p - 1 + Nthreads - 1) / Nthreads;

  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    
    madan <<< Nblocks, Nthreads >>>(d_p, d_g, d_h, d_x);

    hipMemcpy(&x, d_x, sizeof(unsigned int), hipMemcpyDeviceToHost);   
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }
  hipFree(d_p);
  hipFree(d_g);
  hipFree(d_h);
  hipFree(d_x);

  /* Q3 After finding the secret key, decrypt the message */
  int bufferSize = 1024;
  unsigned char *message = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));
  ElGamalDecrypt(Zmessage, a, Nints, p, x);
  convertZToString(Zmessage, Nints, message, Nints * charsPerInt );
  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */
   //declare storage for an ElGamal cryptosytem
  pintf("%s\n", m);
  return 0;
}
