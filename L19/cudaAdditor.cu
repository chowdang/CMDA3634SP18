#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include "hip/hip_runtime.h"

int main(int argc, char **argv) {

  //grt vector size from command line arguement
  int N = atoi(argv[1]);

  //send RNG
  double seed = clock();
  srand48(seed);

  double *h_a, *h_b, *h_c; //host vectors

  h_a = (double *) malloc(N*sizeof(double));
  h_b = (double *) malloc(N*sizeof(double));
  h_c = (double *) malloc(N*sizeof(double));
  
  //duplicata a and b
  for (int n = 0; n < N; n++) {
    h_a[n] = drand48();
    h_b[n] = drand48();
  }
  
  //c = a + b
  for (int n = 0; n < N; n++) {
    h_c[n] = h_a[n] + h_b[n];
  }
  
  double hostEnd = clock();
  double hoseTime (hostEnd - hostStart) / CLOCKS_PER_SEC;
  
  printf("The host clock &g seconds to add a and b \n", hostEnd)

